#include "hip/hip_runtime.h"
#include "add.cuh"

#include <ATen/Dispatch.h>
#include <torch/extension.h>

using torch::Tensor;

template <typename scalar_t>
__global__ void add_kernel(const scalar_t *a, const scalar_t *b, scalar_t *c,
                           size_t size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    c[index] = a[index] + b[index];
  }
}

Tensor ext::add(const Tensor &a, const Tensor &b) {
  auto a_contig = a.contiguous();
  auto b_contig = b.contiguous();
  auto c = torch::empty_like(a);

  const int threads = 1024;
  const int blocks = (a.numel() + threads - 1) / threads;

  AT_DISPATCH_FLOATING_TYPES(a.scalar_type(), "add_cuda", ([&] {
                               add_kernel<scalar_t><<<blocks, threads>>>(
                                   a_contig.const_data_ptr<scalar_t>(),
                                   b_contig.const_data_ptr<scalar_t>(),
                                   c.mutable_data_ptr<scalar_t>(), a.numel());
                             }));

  return c;
}
